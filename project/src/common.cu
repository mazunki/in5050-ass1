#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"

#include <hip/hip_runtime.h>
#include "common.h"

#include <hip/hip_runtime.h>
#include "common.h"

void destroy_frame(frame *f)
{
    if (!f) return;

    // Free CPU memory
    free(f->recons->Y);
    free(f->recons->U);
    free(f->recons->V);
    free(f->recons);

    free(f->predicted->Y);
    free(f->predicted->U);
    free(f->predicted->V);
    free(f->predicted);

    free(f->residuals->Ydct);
    free(f->residuals->Udct);
    free(f->residuals->Vdct);
    free(f->residuals);

    free(f->mbs[Y_COMPONENT]);
    free(f->mbs[U_COMPONENT]);
    free(f->mbs[V_COMPONENT]);

    // Free GPU memory
    hipFree(f->recons->d_Y);
    hipFree(f->recons->d_U);
    hipFree(f->recons->d_V);

    hipFree(f->predicted->d_Y);
    hipFree(f->predicted->d_U);
    hipFree(f->predicted->d_V);

    hipFree(f->residuals->d_Ydct);
    hipFree(f->residuals->d_Udct);
    hipFree(f->residuals->d_Vdct);

    free(f);
}


frame *create_frame(struct c63_common *cm, yuv_t *image)
{
    frame *f = (frame *)malloc(sizeof(frame));
    if (!f) return NULL;

    size_t frame_size = cm->ypw * cm->yph;
    size_t chroma_size = cm->upw * cm->uph;
    size_t num_blocks_Y = cm->mb_rows * cm->mb_cols;
    size_t num_blocks_UV = (cm->mb_rows / 2) * (cm->mb_cols / 2);

    f->orig = image;

    // Allocate CPU memory
    f->recons = (yuv_t *)malloc(sizeof(yuv_t));
    f->recons->Y = (uint8_t *)malloc(frame_size);
    f->recons->U = (uint8_t *)malloc(chroma_size);
    f->recons->V = (uint8_t *)malloc(chroma_size);

    f->predicted = (yuv_t *)malloc(sizeof(yuv_t));
    f->predicted->Y = (uint8_t *)calloc(frame_size, sizeof(uint8_t));
    f->predicted->U = (uint8_t *)calloc(chroma_size, sizeof(uint8_t));
    f->predicted->V = (uint8_t *)calloc(chroma_size, sizeof(uint8_t));

    f->residuals = (dct_t *)malloc(sizeof(dct_t));
    f->residuals->Ydct = (int16_t *)calloc(frame_size, sizeof(int16_t));
    f->residuals->Udct = (int16_t *)calloc(chroma_size, sizeof(int16_t));
    f->residuals->Vdct = (int16_t *)calloc(chroma_size, sizeof(int16_t));

    f->mbs[Y_COMPONENT] = (macroblock *)calloc(num_blocks_Y, sizeof(macroblock));
    f->mbs[U_COMPONENT] = (macroblock *)calloc(num_blocks_UV, sizeof(macroblock));
    f->mbs[V_COMPONENT] = (macroblock *)calloc(num_blocks_UV, sizeof(macroblock));

    // Allocate GPU memory
    hipMalloc((void **)&f->recons->d_Y, frame_size);
    hipMalloc((void **)&f->recons->d_U, chroma_size);
    hipMalloc((void **)&f->recons->d_V, chroma_size);

    hipMalloc((void **)&f->predicted->d_Y, frame_size);
    hipMalloc((void **)&f->predicted->d_U, chroma_size);
    hipMalloc((void **)&f->predicted->d_V, chroma_size);

    hipMalloc((void **)&f->residuals->d_Ydct, frame_size * sizeof(int16_t));
    hipMalloc((void **)&f->residuals->d_Udct, chroma_size * sizeof(int16_t));
    hipMalloc((void **)&f->residuals->d_Vdct, chroma_size * sizeof(int16_t));

    return f;
}

void dump_image(yuv_t *image, int w, int h, FILE *fp) {
	fwrite(image->Y, 1, w*h, fp);
	fwrite(image->U, 1, w*h/4, fp);
	fwrite(image->V, 1, w*h/4, fp);
}

