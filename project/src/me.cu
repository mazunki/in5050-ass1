#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

#include "me.h"
#include "tables.h"
#include "common.h"

#define MACROBLOCK_SIZE 8
#define CUDA_THREADS_PER_BLOCK_X 16
#define CUDA_THREADS_PER_BLOCK_Y 16

__device__ static int sad_block_8x8(uint8_t *block1, uint8_t *block2, int stride)
{
  int u, v;
  int result = 0;
  for (v = 0; v < MACROBLOCK_SIZE; ++v)
  {
    for (u = 0; u < MACROBLOCK_SIZE; ++u)
    {
      result += abs(block2[v*stride+u] - block1[v*stride+u]);
    }
  }
  return result;
}

/* Motion estimation for an 8x8 block */
__device__ static void me_block_8x8(struct macroblock *mb, int mb_x, int mb_y,
                                    uint8_t *orig, uint8_t *ref, int padw, int padh, int range)
{
  /* Make sure we are within bounds of reference frame. TODO: Support partial
     frame bounds. */
  int left   = MAX(mb_x * MACROBLOCK_SIZE - range, 0);
  int top    = MAX(mb_y * MACROBLOCK_SIZE - range, 0);
  int right  = MIN(mb_x * MACROBLOCK_SIZE + range, padw - MACROBLOCK_SIZE);
  int bottom = MIN(mb_y * MACROBLOCK_SIZE + range, padh - MACROBLOCK_SIZE);

  int x, y;
  int mx = mb_x * MACROBLOCK_SIZE;
  int my = mb_y * MACROBLOCK_SIZE;
  int best_sad = INT_MAX;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      int sad = sad_block_8x8(orig + my*padw + mx, ref + y*padw + x, padw);
      if (sad < best_sad)
      {
        mb->mv_x = x - mx;
        mb->mv_y = y - my;
        best_sad = sad;
      }
    }
  }

  /* Here, there should be a threshold on SAD that checks if the motion vector
     is cheaper than intraprediction. We always assume MV to be beneficial */

  /* printf("Using motion vector (%d, %d) with SAD %d\n", mb->mv_x, mb->mv_y,
     best_sad); */

  mb->use_mv = 1;
}

__global__ void c63_motion_estimate_kernel(uint8_t *d_orig, uint8_t *d_recons, macroblock *d_mbs, int width, int height, int range) {
  int mb_x = blockIdx.x * blockDim.x + threadIdx.x;
  int mb_y = blockIdx.y * blockDim.y + threadIdx.y;
  int mb_cols = width / MACROBLOCK_SIZE;
  int mb_rows = height / MACROBLOCK_SIZE;

  if (mb_x >= mb_cols || mb_y >= mb_rows) {
    return;
  }

  macroblock *mb = &d_mbs[mb_y * mb_cols + mb_x];
  me_block_8x8(mb, mb_x, mb_y, d_orig, d_recons, width, height, range);
}

__host__ void c63_motion_estimate(struct c63_common *cm)
{
  /* Compare this frame with previous reconstructed frame */
  int range = cm->me_search_range;
  size_t frame_size = cm->ypw * cm->yph;
  size_t chroma_size = (cm->ypw / 2) * (cm->yph / 2);
  size_t num_blocks_luma = cm->mb_rows * cm->mb_cols;
  size_t num_blocks_chroma = (cm->mb_rows / 2) * (cm->mb_cols / 2);

  dim3 block_size(CUDA_THREADS_PER_BLOCK_X, CUDA_THREADS_PER_BLOCK_Y);
  dim3 grid_size(cm->padw[Y_COMPONENT] / MACROBLOCK_SIZE, cm->padh[Y_COMPONENT] / MACROBLOCK_SIZE);

  CUDA_CHECK(hipMemcpy(cm->curframe->orig->d_Y, cm->curframe->orig->Y, frame_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(cm->curframe->orig->d_U, cm->curframe->orig->U, chroma_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(cm->curframe->orig->d_V, cm->curframe->orig->V, chroma_size, hipMemcpyHostToDevice));

  CUDA_CHECK(hipMemcpy(cm->refframe->recons->d_Y, cm->refframe->recons->Y, frame_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(cm->refframe->recons->d_U, cm->refframe->recons->U, chroma_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(cm->refframe->recons->d_V, cm->refframe->recons->V, chroma_size, hipMemcpyHostToDevice));

  /* Luma */
  c63_motion_estimate_kernel<<<grid_size, block_size>>>(cm->curframe->orig->d_Y, cm->refframe->recons->d_Y, cm->curframe->d_mbs[Y_COMPONENT], cm->padw[Y_COMPONENT], cm->padh[Y_COMPONENT], range);
  CUDA_ASSERT();

  /* Chroma */
  c63_motion_estimate_kernel<<<grid_size, block_size>>>(cm->curframe->orig->d_U, cm->refframe->recons->d_U, cm->curframe->d_mbs[U_COMPONENT], cm->padw[U_COMPONENT], cm->padh[U_COMPONENT], range/2);
  CUDA_ASSERT();

  c63_motion_estimate_kernel<<<grid_size, block_size>>>(cm->curframe->orig->d_V, cm->refframe->recons->d_V, cm->curframe->d_mbs[V_COMPONENT], cm->padw[V_COMPONENT], cm->padh[V_COMPONENT], range/2);
  CUDA_ASSERT();

  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(cm->curframe->mbs[Y_COMPONENT], cm->curframe->d_mbs[Y_COMPONENT], num_blocks_luma * sizeof(struct macroblock), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(cm->curframe->mbs[U_COMPONENT], cm->curframe->d_mbs[U_COMPONENT], num_blocks_chroma * sizeof(struct macroblock), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(cm->curframe->mbs[V_COMPONENT], cm->curframe->d_mbs[V_COMPONENT], num_blocks_chroma * sizeof(struct macroblock), hipMemcpyDeviceToHost));
}

/* Motion compensation for 8x8 block */
static void mc_block_8x8(struct macroblock *mb, int mb_x, int mb_y,
                         uint8_t *predicted, uint8_t *ref, int padw)
{
  if (!mb->use_mv) { return; }

  int left = mb_x * MACROBLOCK_SIZE;
  int top = mb_y * MACROBLOCK_SIZE;
  int right = left + MACROBLOCK_SIZE;
  int bottom = top + MACROBLOCK_SIZE;
  int w = padw;

  for (int y = top; y < bottom; ++y)
  {
    for (int x = left; x < right; ++x)
    {
      predicted[y * w + x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
    }
  }
}

/* Motion compensation kernel function (still CPU-based) */
static void c63_motion_compensate_kernel(struct macroblock *mbs, int mb_cols, int mb_rows,
                                         uint8_t *predicted, uint8_t *ref, int padw)
{
  for (int mb_y = 0; mb_y < mb_rows; ++mb_y)
  {
    for (int mb_x = 0; mb_x < mb_cols; ++mb_x)
    {
      struct macroblock *mb = &mbs[mb_y * mb_cols + mb_x];

      mc_block_8x8(mb, mb_x, mb_y, predicted, ref, padw);
    }
  }
}

void c63_motion_compensate(struct c63_common *cm)
{
  /* Luma */
  c63_motion_compensate_kernel(cm->curframe->mbs[Y_COMPONENT], cm->mb_cols, cm->mb_rows, cm->curframe->predicted->Y, cm->refframe->recons->Y, cm->padw[Y_COMPONENT]);

  /* Chroma */
  c63_motion_compensate_kernel(cm->curframe->mbs[U_COMPONENT], cm->mb_cols / 2, cm->mb_rows / 2, cm->curframe->predicted->U, cm->refframe->recons->U, cm->padw[U_COMPONENT]);

  c63_motion_compensate_kernel(cm->curframe->mbs[V_COMPONENT], cm->mb_cols / 2, cm->mb_rows / 2, cm->curframe->predicted->V, cm->refframe->recons->V, cm->padw[V_COMPONENT]);
}
