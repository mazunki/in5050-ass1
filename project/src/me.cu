#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>

#include "me.h"
#include "tables.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__device__ int sad_block_8x8(uint8_t *orig, uint8_t *ref, int w) {
    int sad = 0;
    for (int i = 0; i < 8; ++i) {
        for (int j = 0; j < 8; ++j) {
            sad += abs(orig[i * w + j] - ref[i * w + j]);
        }
    }
    return sad;
}

__global__ void me_block_8x8_cuda(uint8_t *orig, uint8_t *ref, int w, int h,
                                  int mb_x, int mb_y, int range, int *best_mv_x, int *best_mv_y) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int mx = mb_x * 8;
    int my = mb_y * 8;

    int left = max(0, mx - range);
    int top = max(0, my - range);
    int right = min(w - 8, mx + range);
    int bottom = min(h - 8, my + range);

    int search_x = left + bx * BLOCK_SIZE_X + tx;
    int search_y = top + by * BLOCK_SIZE_Y + ty;

    if (search_x >= right || search_y >= bottom) return;

    __shared__ int best_sad[BLOCK_SIZE_X * BLOCK_SIZE_Y];
    __shared__ int best_x[BLOCK_SIZE_X * BLOCK_SIZE_Y];
    __shared__ int best_y[BLOCK_SIZE_X * BLOCK_SIZE_Y];

    int tid = ty * BLOCK_SIZE_X + tx;
    best_sad[tid] = INT_MAX;

    if (search_x < right && search_y < bottom) {
        int sad = sad_block_8x8(orig + my * w + mx, ref + search_y * w + search_x, w);
        best_sad[tid] = sad;
        best_x[tid] = search_x - mx;
        best_y[tid] = search_y - my;
    }

    __syncthreads();

    // Parallel reduction within the block
    if (tid == 0) {
        int min_sad = INT_MAX;
        int min_x = 0, min_y = 0;
        for (int i = 0; i < BLOCK_SIZE_X * BLOCK_SIZE_Y; i++) {
            if (best_sad[i] < min_sad) {
                min_sad = best_sad[i];
                min_x = best_x[i];
                min_y = best_y[i];
            }
        }
        atomicMin(best_mv_x, min_x);
        atomicMin(best_mv_y, min_y);
    }
}

void me_block_8x8(struct c63_common *cm, int mb_x, int mb_y,
    uint8_t *orig, uint8_t *ref, int color_component) {
    int range = cm->me_search_range;
    if (color_component > 0) { range /= 2; }

    int w = cm->padw[color_component];
    int h = cm->padh[color_component];

    int *d_best_mv_x, *d_best_mv_y;
    hipMalloc(&d_best_mv_x, sizeof(int));
    hipMalloc(&d_best_mv_y, sizeof(int));
    hipMemset(d_best_mv_x, 0, sizeof(int));
    hipMemset(d_best_mv_y, 0, sizeof(int));

    dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 gridSize((range * 2 + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X,
                  (range * 2 + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y);

    me_block_8x8_cuda<<<gridSize, blockSize>>>(orig, ref, w, h, mb_x, mb_y, range, d_best_mv_x, d_best_mv_y);

    int best_mv_x, best_mv_y;
    hipMemcpy(&best_mv_x, d_best_mv_x, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&best_mv_y, d_best_mv_y, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_best_mv_x);
    hipFree(d_best_mv_y);

    struct macroblock *mb =
        &cm->curframe->mbs[color_component][mb_y * cm->padw[color_component] / 8 + mb_x];

    mb->mv_x = best_mv_x;
    mb->mv_y = best_mv_y;
    mb->use_mv = 1;
}


void c63_motion_estimate(struct c63_common *cm)
{
  /* Compare this frame with previous reconstructed frame */
  int mb_x, mb_y;

  /* Luma */
  for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
    {
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->Y,
          cm->refframe->recons->Y, Y_COMPONENT);
    }
  }

  /* Chroma */
  for (mb_y = 0; mb_y < cm->mb_rows / 2; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols / 2; ++mb_x)
    {
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->U,
          cm->refframe->recons->U, U_COMPONENT);
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->V,
          cm->refframe->recons->V, V_COMPONENT);
    }
  }
}

/* Motion compensation for 8x8 block */
static void mc_block_8x8(struct c63_common *cm, int mb_x, int mb_y,
    uint8_t *predicted, uint8_t *ref, int color_component)
{
  struct macroblock *mb =
    &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  if (!mb->use_mv) { return; }

  int left = mb_x * 8;
  int top = mb_y * 8;
  int right = left + 8;
  int bottom = top + 8;

  int w = cm->padw[color_component];

  /* Copy block from ref mandated by MV */
  int x, y;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
    }
  }
}

void c63_motion_compensate(struct c63_common *cm)
{
  int mb_x, mb_y;

  /* Luma */
  for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->Y,
          cm->refframe->recons->Y, Y_COMPONENT);
    }
  }

  /* Chroma */
  for (mb_y = 0; mb_y < cm->mb_rows / 2; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols / 2; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->U,
          cm->refframe->recons->U, U_COMPONENT);
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->V,
          cm->refframe->recons->V, V_COMPONENT);
    }
  }
}
