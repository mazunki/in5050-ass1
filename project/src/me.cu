#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

#include "me.h"
#include "tables.h"
#include "common.h"

#define MACROBLOCK_SIZE 8
#define CUDA_THREADS_PER_BLOCK_X 16
#define CUDA_THREADS_PER_BLOCK_Y 16

__device__ static int sad_block_8x8(uint8_t *block1, uint8_t *block2, int stride)
{
  int u, v;

  int result = 0;

  for (v = 0; v < MACROBLOCK_SIZE; ++v)
  {
    for (u = 0; u < MACROBLOCK_SIZE; ++u)
    {
      result += abs(block2[v*stride+u] - block1[v*stride+u]);
    }
  }

  return result;
}

/* Motion estimation for 8x8 block */
__device__ static void me_block_8x8(struct macroblock *mb, int mb_x, int mb_y,
    uint8_t *orig, uint8_t *ref, int padw, int padh, int range)
{
  /* Make sure we are within bounds of reference frame. TODO: Support partial
     frame bounds. */
  int left   = MAX(mb_x * MACROBLOCK_SIZE - range, 0);
  int top    = MAX(mb_y * MACROBLOCK_SIZE - range, 0);
  int right  = MIN(mb_x * MACROBLOCK_SIZE + range, padw - MACROBLOCK_SIZE);
  int bottom = MIN(mb_y * MACROBLOCK_SIZE + range, padh - MACROBLOCK_SIZE);

  int x, y;

  int mx = mb_x * MACROBLOCK_SIZE;
  int my = mb_y * MACROBLOCK_SIZE;

  int best_sad = INT_MAX;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      int sad = sad_block_8x8(orig + my*padw+mx, ref + y*padw+x, padw);

      /* printf("(%4d,%4d) - %d\n", x, y, sad); */

      if (sad < best_sad)
      {
        mb->mv_x = x - mx;
        mb->mv_y = y - my;
        best_sad = sad;
      }
    }
  }

  /* Here, there should be a threshold on SAD that checks if the motion vector
     is cheaper than intraprediction. We always assume MV to be beneficial */

  /* printf("Using motion vector (%d, %d) with SAD %d\n", mb->mv_x, mb->mv_y,
     best_sad); */

  mb->use_mv = 1;
}

__global__ void c63_motion_estimate_kernel(uint8_t *d_orig, uint8_t *d_recons, macroblock *d_mbs, int width, int height, int range) {
  for (int mb_y = 0; mb_y < height / MACROBLOCK_SIZE; ++mb_y) {
    for (int mb_x = 0; mb_x < height / MACROBLOCK_SIZE; ++mb_x) {
      macroblock *mb = &d_mbs[mb_y * (width / MACROBLOCK_SIZE) + mb_x];
      me_block_8x8(mb, mb_x, mb_y, d_orig, d_recons, width, height, range);
    }
  }
}

__host__ void c63_motion_estimate(struct c63_common *cm)
{
  /* Compare this frame with previous reconstructed frame */
  int range = cm->me_search_range;

  dim3 block_size(CUDA_THREADS_PER_BLOCK_X, CUDA_THREADS_PER_BLOCK_Y);
  dim3 grid_size(cm->padw[Y_COMPONENT]/MACROBLOCK_SIZE, cm->padh[Y_COMPONENT]/MACROBLOCK_SIZE);

  /* Luma */
  c63_motion_estimate_kernel<<<grid_size, block_size>>>(cm->curframe->orig->Y, cm->refframe->recons->Y, cm->curframe->mbs[Y_COMPONENT], cm->padw[Y_COMPONENT], cm->padh[Y_COMPONENT], range);
  CUDA_CHECK(hipDeviceSynchronize());

  /* Chroma */
  c63_motion_estimate_kernel<<<grid_size, block_size>>>(cm->curframe->orig->U, cm->refframe->recons->U, cm->curframe->mbs[U_COMPONENT], cm->padw[U_COMPONENT], cm->padh[U_COMPONENT], range);
  CUDA_CHECK(hipDeviceSynchronize());

  c63_motion_estimate_kernel<<<grid_size, block_size>>>(cm->curframe->orig->V, cm->refframe->recons->V, cm->curframe->mbs[V_COMPONENT], cm->padw[V_COMPONENT], cm->padh[V_COMPONENT], range);
  CUDA_CHECK(hipDeviceSynchronize());
}

/* Motion compensation for 8x8 block */
static void mc_block_8x8(struct c63_common *cm, int mb_x, int mb_y,
    uint8_t *predicted, uint8_t *ref, int color_component)
{
  struct macroblock *mb =
    &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/MACROBLOCK_SIZE+mb_x];

  if (!mb->use_mv) { return; }

  int left = mb_x * MACROBLOCK_SIZE;
  int top = mb_y * MACROBLOCK_SIZE;
  int right = left + MACROBLOCK_SIZE;
  int bottom = top + MACROBLOCK_SIZE;

  int w = cm->padw[color_component];

  /* Copy block from ref mandated by MV */
  int x, y;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
    }
  }
}

void c63_motion_compensate(struct c63_common *cm)
{
  int mb_x, mb_y;

  /* Luma */
  for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->Y,
          cm->refframe->recons->Y, Y_COMPONENT);
    }
  }

  /* Chroma */
  for (mb_y = 0; mb_y < cm->mb_rows / 2; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols / 2; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->U,
          cm->refframe->recons->U, U_COMPONENT);
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->V,
          cm->refframe->recons->V, V_COMPONENT);
    }
  }
}
